#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "../ImageWriter/ImageWriter.h"
#include "../ImageReader/ImageReader.h"
#include "../Packed3DArray/Packed3DArray.h"
using namespace std;

int main(int argc, char *argv[]){

    ImageReader* ir = ImageReader::create("../Images/door.jpg");
    if (ir == nullptr) exit(1);

	cryph::Packed3DArray<unsigned char>* rgb = ir->getInternalPacked3DArrayImage();
	int xres = ir->getWidth();
	int yres = ir->getHeight();
    int size = xres*yres;
    unsigned char* rChannelH = new unsigned char[size];
    unsigned char* gChannelH = new unsigned char[size];
    unsigned char* bChannelH = new unsigned char[size];
	for (int r=0 ; r<yres ; r++)
		for (int c=0 ; c<xres ; c++)
		{
			unsigned char R = rgb->getDataElement(r, c, 0);
			unsigned char G = rgb->getDataElement(r, c, 1);
			unsigned char B = rgb->getDataElement(r, c, 2);
			// (This simple demo does not modify the alpha channel, if present.)
			// Circular shift the colors:
            rChannelH[r*c] = R;
            gChannelH[r*c] = G;
            bChannelH[r*c] = B;
		}
	
    unsigned char* rChannelD;
    hipMalloc((void**)&rChannelD, size);
    unsigned char* gChannelD;
    hipMalloc((void**)&gChannelD, size);
    unsigned char* bChannelD;
    hipMalloc((void**)&bChannelD, size);
    hipMemcpy(rChannelD, rChannelH, size, hipMemcpyHostToDevice);
	hipMemcpy(gChannelD, gChannelH, size, hipMemcpyHostToDevice);
    hipMemcpy(bChannelD, bChannelH, size, hipMemcpyHostToDevice);



    ifstream filterFile;
    filterFile.open(argv[2]);
    float x;
    while(filterFile >> x){

    }

  //  hipMalloc((void**)&d_X, size);

}